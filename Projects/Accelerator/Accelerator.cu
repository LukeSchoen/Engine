#include "hip/hip_runtime.h"
#include "Accelerator.h"

// System includes
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>
// CUDA Helper functions

#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include ""

__global__ void matrixMulCUDA(float *input, float *output, float *matrix)
{
  int ID = (blockIdx.x * blockDim.x + threadIdx.x);
  //output[ID] = input[ID] + 1;
}

Accelerator::Accelerator()
{
  int devID = 0;

  hipDeviceProp_t deviceProp;
  assert(!hipGetDevice(&devID));
  assert(!hipGetDeviceProperties(&deviceProp, devID));
  printf("Using GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

  printf("integrated %d\n", deviceProp.integrated);
  printf("totalGlobalMem %d\n", deviceProp.totalGlobalMem);
  printf("clockRate %d\n", deviceProp.clockRate);
  printf("multiProcessorCount %d\n", deviceProp.multiProcessorCount);
  printf("ThreadsPerMultiProcessor %d\n", deviceProp.maxThreadsPerMultiProcessor);
  printf("ThreadsPerBlock%d \n", deviceProp.maxThreadsPerBlock);
  printf("warpSize %d\n", deviceProp.warpSize);
  printf("regsPerBlock %d\n", deviceProp.regsPerBlock);

  // Use a larger block size for Fermi and above
  block_size = (deviceProp.major < 2) ? 16 : 32;
  multiProcessors = deviceProp.multiProcessorCount;
  maxThreadsPerMultiProcessor = deviceProp.maxThreadsPerMultiProcessor;
}
 
int Accelerator::OptimalThreadCount()
{
  return multiProcessors * maxThreadsPerMultiProcessor;
}

void Accelerator::MatVec(const float *input, float *output, const float *matrix, int64_t numVerts)
{
  // Allocate device memory
  float *inputGPU;
  float *outputGPU;
  float *matrixGPU;
  assert(!hipMalloc(&inputGPU, numVerts * 3 * sizeof(float)));
  assert(!hipMalloc(&outputGPU, numVerts * 3 * sizeof(float)));
  assert(!hipMalloc(&matrixGPU, 16 * sizeof(float)));

  // copy host memory to device
  assert(!hipMemcpy(matrixGPU, matrix, sizeof(float) * 16, hipMemcpyHostToDevice));
  assert(!hipMemcpy(inputGPU, input, sizeof(float) * 16, hipMemcpyHostToDevice));

  // Setup execution parameters
  matrixMulCUDA<<<numVerts / 256, 256>>>(matrixGPU, inputGPU, outputGPU);

  //hipDeviceSynchronize();

  assert(!hipMemcpy(output, outputGPU, numVerts * 3 * sizeof(float), hipMemcpyDeviceToHost));

  // Clean up memory
  hipFree(matrixGPU);
  hipFree(inputGPU);
  hipFree(outputGPU);
}
