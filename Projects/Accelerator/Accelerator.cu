#include "hip/hip_runtime.h"
#include "Accelerator.h"

// System includes
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include ""

__global__ void matrixMulCUDA(float *matrix, float *input, float *output)
{
  int ID = (blockIdx.x * blockDim.x + threadIdx.x) * 3;
  output[ID + 0] = input[ID] * matrix[0] + input[ID + 1] * matrix[1] + input[ID + 2] * matrix[2] + 1 * matrix[3];
  output[ID + 1] = input[ID] * matrix[4] + input[ID + 1] * matrix[5] + input[ID + 2] * matrix[6] + 1 * matrix[7];
  output[ID + 2] = input[ID] * matrix[8] + input[ID + 1] * matrix[9] + input[ID + 2] * matrix[10] + 1 * matrix[11];
}

Accelerator::Accelerator()
{
  int devID = 0;

  hipDeviceProp_t deviceProp;
  assert(!hipGetDevice(&devID));
  assert(!hipGetDeviceProperties(&deviceProp, devID));
  printf("Using GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

  printf("integrated %d\n", deviceProp.integrated);
  printf("totalGlobalMem %d\n", deviceProp.totalGlobalMem);
  printf("clockRate %d\n", deviceProp.clockRate);
  printf("multiProcessorCount %d\n", deviceProp.multiProcessorCount);
  printf("ThreadsPerMultiProcessor %d\n", deviceProp.maxThreadsPerMultiProcessor);
  printf("ThreadsPerBlock%d \n", deviceProp.maxThreadsPerBlock);
  printf("warpSize %d\n", deviceProp.warpSize);
  printf("regsPerBlock %d\n", deviceProp.regsPerBlock);

  // Use a larger block size for Fermi and above
  block_size = (deviceProp.major < 2) ? 16 : 32;
  multiProcessors = deviceProp.multiProcessorCount;
  maxThreadsPerMultiProcessor = deviceProp.maxThreadsPerMultiProcessor;
}
 
int Accelerator::OptimalThreadCount()
{
  return multiProcessors * maxThreadsPerMultiProcessor;
}

void Accelerator::MatVec(const float *input, float *output, const float *matrix, int64_t numVerts)
{
  // Allocate device memory
  float *inputGPU;
  float *outputGPU;
  float *matrixGPU;
  assert(!hipMalloc(&inputGPU, numVerts * 3 * sizeof(float)));
  assert(!hipMalloc(&outputGPU, numVerts * 3 * sizeof(float)));
  assert(!hipMalloc(&matrixGPU, 16 * sizeof(float)));

  // copy host memory to device
  assert(!hipMemcpy(matrixGPU, matrix, sizeof(float) * 16, hipMemcpyHostToDevice));
  assert(!hipMemcpy(inputGPU, input, numVerts * 3 * sizeof(float), hipMemcpyHostToDevice));

  // Setup execution parameters
  for (int64_t i = 0; i < 1; i++)
    hipError_t code = matrixMulCUDA<<<numVerts / 256, 256>>>(matrixGPU, inputGPU, outputGPU);
    if(code != hipSuccess)
    {

    }
  

if ()
{
  std::stringstream ss;
  ss << file << "(" << line << ")";
  std::string file_and_line;
  ss >> file_and_line;
  throw thrust::system_error(code, thrust::cuda_category(), file_and_line);
}

  //hipDeviceSynchronize();

  assert(!hipMemcpy(output, outputGPU, numVerts * 3 * sizeof(float), hipMemcpyDeviceToHost));

  // Clean up memory
  hipFree(matrixGPU);
  hipFree(inputGPU);
  hipFree(outputGPU);
}
